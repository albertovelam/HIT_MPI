#include "turH.h"

void kernelCheck( hipError_t error, const char* function, int a=1)
{
	if(error !=hipSuccess)
	{
		const char* error_string= hipGetErrorString(error);
		printf("\n error  %s : %s \n", function, error_string);
		exit(1);
	}
		
	
	if(a!=0)
	{
	error= hipGetLastError();			
		if(error !=hipSuccess)
		{
			const char* error_string= hipGetErrorString(error);
			printf("\n error  %s : %s \n", function, error_string);
			exit(1);
		}
	}

	return;
}



